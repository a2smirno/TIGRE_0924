#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for Steepest descend in POCS-type algorithms.
 *
 * This file will iteratively minimize by stepest descend the total variation
 * of the input image, with the parameters given, using GPUs.
 *
 * CODE by       Ander Biguri
 *
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */







#define MAXTHREADS 1024
#define MAX_BUFFER 60

#include "POCS_TV.hpp"




#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                hipDeviceReset();\
                mexErrMsgIdAndTxt("CBCT:CUDA:POCS_TV",hipGetErrorString(__err));\
        } \
} while (0)
    
// CUDA kernels
//https://stackoverflow.com/questions/21332040/simple-cuda-kernel-optimization/21340927#21340927
    __global__ void divideArrayScalar(float* vec,float scalar,const size_t n){
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]/=scalar;
        }
    }
    __global__ void multiplyArrayScalar(float* vec,float scalar,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]*=scalar;
        }
    }
    __global__ void substractArrays(float* vec,float* vec2,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]-=vec2[i];
        }
    }
    
    __device__ __inline__
            void gradient(const float* u, float* grad,
            long z, long y, long x,
            long depth, long rows, long cols){
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;
        
        float uidx = u[idx];
        
        if ( z - 1 >= 0 && z<depth) {
            grad[0] = (uidx-u[(z-1)*size2d + y*cols + x]) ;
        }
        
        if ( y - 1 >= 0 && y<rows){
            grad[1] = (uidx-u[z*size2d + (y-1)*cols + x]) ;
        }
        
        if ( x - 1 >= 0 && x<cols) {
            grad[2] = (uidx-u[z*size2d + y*cols + (x-1)]);
        }
    }
    __device__ __inline__
            void prox_mu_sigma_g(float* u, const float* Lop, const float sgm, const float mu){
        
        float t[6] = {0.f,0.f,0.f,0.f,0.f,0.f};
        float musgm = mu*sgm;

        t[0] = u[0]-sgm*Lop[0]-(u[0]-sgm*Lop[0])/fmaxf(musgm*sqrtf(u[0]*u[0]+u[1]*u[1]),1);
        t[1] = u[1]-sgm*Lop[1]-(u[1]-sgm*Lop[1])/fmaxf(musgm*sqrtf(u[0]*u[0]+u[1]*u[1]),1);
        t[2] = u[2]-sgm*Lop[2]-(u[2]-sgm*Lop[2])/fmaxf(musgm*sqrtf(u[2]*u[2]+u[3]*u[3]),1);
        t[3] = u[3]-sgm*Lop[3]-(u[3]-sgm*Lop[3])/fmaxf(musgm*sqrtf(u[2]*u[2]+u[3]*u[3]),1);
        t[4] = u[4]-sgm*Lop[4]-(u[4]-sgm*Lop[4])/fmaxf(musgm*sqrtf(u[4]*u[4]+u[5]*u[5]),1);
        t[5] = u[5]-sgm*Lop[5]-(u[5]-sgm*Lop[5])/fmaxf(musgm*sqrtf(u[4]*u[4]+u[5]*u[5]),1);

        u[0] = t[0];
        u[1] = t[1];
        u[2] = t[2];
        u[3] = t[3];
        u[4] = t[4];
        u[5] = t[5];

    }
        // Define the opL function as a device inline function
    __device__ __inline__ void opL(const float* df, const float* dfi, const float* dfj,
            const float* dfl, const float* dfk, const float* dfp, const float* dfq,
            const float* vadj, const float* vadj_i, const float* vadj_j, 
            const float* vadj_l, const float* vadj_k, const float* vadj_p, const float* vadj_q,
            const float* f1, const float* f2, float mu, float* t,
            long z, long y, long x,
            long depth, long rows, long cols){
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;

        float u1 = df[0]+vadj[0]-f1[idx]/mu;
        float u2 = df[0]+vadj[1]-f2[idx]/mu;
        float u1i = 0.f;
        float u2j = 0.f;
        float u1l = 0.f;
        float u2k = 0.f;
        float u1p = 0.f;
        float u2q = 0.f;

        if (x - 1 >= 0 && x<cols) {
            u1i = dfi[0]+vadj_i[0]-f1[idx-1]/mu;
        }
        if (y - 1 >= 0 && y<rows) {
            u2j = dfj[1]+vadj_j[1]-f2[idx-cols]/mu;
        }
        if (y >= 0 && y<rows-1) {
            u1l = dfl[0]+vadj_l[0]-f1[idx+cols]/mu;
        }
        if (x >= 0 && x<cols-1) {
            u2k = dfk[1]+vadj_k[1]-f2[idx+1]/mu;
        }
        if (y >= 0 && y<rows-1 && x - 1 >= 0 && x<cols) {
            u1p = dfp[0]+vadj_p[0]-f1[idx-1+cols]/mu;
        }
        if (x >= 0 && x<cols-1 && y - 1 >= 0 && y<rows) {
            u2q = dfq[1]+vadj_q[1]-f2[idx+1-cols]/mu;
        }

        t[0] = u1; //df[0]+vadj[0]-f1[idx]/mu
        t[1] = 0.25f*(u2+u2j+u2k+u2q);
        t[2] = 0.25f*(u1+u1i+u1l+u1p);
        t[3] = u2;
        t[4] = 0.5f*(u1+u1i);
        t[5] = 0.5f*(u2+u2j);
    }

    __device__ __inline__ void opLadj(const float* u, const float* u11, const float* u12, const float* u21, 
            const float* u22, const float* u31, const float* u32, float* t,
            long z, long y, long x,
            long depth, long rows, long cols){
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;
        
        t[0] = u[0];
        t[1] = u[3];

        t[0] += u21[idx]*0.25f;
        t[1] += u12[idx]*0.25f;

        if (y - 1 >= 0 && y<rows) {
            t[0] += u21[idx - cols]*0.25f;
        }
        if (y >= 0 && y<rows-1) {
            t[1] += u12[idx + cols]*0.25f;
        }
        if (x >= 0 && x<cols-1) {
            t[0] += u21[idx + 1]*0.25f;
        }
        if (x - 1 >= 0 && x<cols) {
            t[1] += u12[idx - 1]*0.25f;
        }
        if (y - 1 >= 0 && y<rows && x >= 0 && x<cols-1) {
            t[0] += u21[idx + 1 - cols]*0.25f;
        }
        if (x - 1 >= 0 && x<cols && y >= 0 && y<rows-1) {
            t[1] += u12[idx - 1 + cols]*0.25f;
        }
        
        t[0] += u31[idx]*0.5f;
        t[1] += u32[idx]*0.5f;

        if (x >= 0 && x<cols-1) {
            t[0] += u31[idx + 1]*0.5f;
        }
        if (y >= 0 && y<rows-1) {
            t[1] += u32[idx + cols]*0.5f;
        }
    }

    __global__ void APGM(const float* f, float* f1, float* f2, float* v11, float* v21, float* v31, float* v12, float* v22, float* v32,
            long depth, long rows, long cols,const float delta, const float mu0, int maxIter){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        if ( x >= cols || y >= rows || z >= depth )
            return;
        
        float sgm = 0.33f;
        float mu = mu0;
        float v[6]     = {0.f,0.f,0.f,0.f,0.f,0.f};
        float vadj[2]  = {0.f,0.f};
        float vadjx[2] = {0.f,0.f};
        float vadjy[2] = {0.f,0.f};
        float vadjl[2] = {0.f,0.f};
        float vadjk[2] = {0.f,0.f};
        float vadjp[2] = {0.f,0.f};
        float vadjq[2] = {0.f,0.f};
        float Lop[6]   = {0.f,0.f,0.f,0.f,0.f,0.f};

        float df[3]  ={0.f,0.f,0.f};
        float dfi[3] ={0.f,0.f,0.f};
        float dfj[3] ={0.f,0.f,0.f};
        float dfl[3] ={0.f,0.f,0.f};
        float dfk[3] ={0.f,0.f,0.f};
        float dfp[3] ={0.f,0.f,0.f};
        float dfq[3] ={0.f,0.f,0.f};

        //need Du only at one point to intialize the loop to find the dual test function
        gradient(f,df   ,z  ,y   ,x  , depth,rows,cols);
        gradient(f,dfi  ,z  ,y   ,x-1 , depth,rows,cols);
        gradient(f,dfj  ,z  ,y-1 ,x  , depth,rows,cols);
        gradient(f,dfl  ,z  ,y   ,x+1 , depth,rows,cols);
        gradient(f,dfk  ,z  ,y-1 ,x+1  , depth,rows,cols);
        gradient(f,dfp  ,z  ,y+1 ,x   , depth,rows,cols);
        gradient(f,dfq  ,z  ,y+1 ,x-1  , depth,rows,cols);

        //iitialize vector field
        v[0] = v11[idx];
        v[1] = v12[idx];
        v[2] = v21[idx];
        v[3] = v22[idx];
        v[4] = v31[idx];
        v[5] = v32[idx];
        vadj[0] = v[0];
        vadj[1] = v[3];

        //APGM loop
        for(int i = 0; i < maxIter; i++){
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjx  ,z  ,y   ,x-1 , depth,rows,cols);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjy  ,z  ,y-1 ,x  , depth,rows,cols);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjl  ,z  ,y   ,x+1 , depth,rows,cols);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjk  ,z  ,y-1 ,x+1  , depth,rows,cols);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjp  ,z  ,y+1 ,x , depth,rows,cols);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadjq  ,z  ,y+1 ,x-1 , depth,rows,cols);
            opL(df, dfi, dfj, dfl, dfk, dfp, dfq, vadj, vadjx, vadjy, vadjl, vadjk, vadjp, vadjq, f1, f2, mu, Lop, z  ,y  ,x  , depth,rows,cols); 
            prox_mu_sigma_g(v,Lop,sgm,mu);
            opLadj(v, v11, v12, v21, v22, v31, v32, vadj  ,z  ,y  ,x  , depth,rows,cols);
            f1[idx] = f2[idx]-(-df[0]+vadj[0])*mu;
            f2[idx] = f2[idx]-(-df[1]+vadj[1])*mu;
            mu *= 0.99f;
        }
        return;
        
    }

    __global__ void gradientDTV(const float* f1, const float* f2, float* dftv,
            long depth, long rows, long cols,const float delta){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        unsigned long long idy = z * rows * cols + y * cols + x-1;
        unsigned long long idx2 = z * rows * cols + (y-1) * cols + x;
        if ( x -1 <= 0 || y-1 <= 0 || z > depth )
            return;
        
        dftv[idx]=-f1[idx]+f1[idx2]-f2[idx]+f2[idy];
        
        return;
        
    }

    __global__ void gradientTV(const float* f, float* dftv,
            long depth, long rows, long cols,const float delta){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        if ( x >= cols || y >= rows || z >= depth )
            return;
        
        
        float df[3] ={0.f,0.f,0.f};
        float dfi[3]={0.f,0.f,0.f}; // dfi== \partial f_{i+1,j,k}
        float dfj[3]={0.f,0.f,0.f};
        float dfk[3]={0.f,0.f,0.f};
        gradient(f,df  ,z  ,y  ,x  , depth,rows,cols);
        gradient(f,dfi ,z  ,y  ,x+1, depth,rows,cols);
        gradient(f,dfj ,z  ,y+1,x  , depth,rows,cols);
        gradient(f,dfk ,z+1,y  ,x  , depth,rows,cols);
        float eps=0.00000001; //% avoid division by zero
        
        float wx=__expf(-(df[0]/delta)*(df[0]/delta));
        float wy=__expf(-(df[1]/delta)*(df[1]/delta));
        float wz=__expf(-(df[2]/delta)*(df[2]/delta));
        
        float wxi=__expf(-(dfi[0]/delta)*(dfi[0]/delta));
        float wyi=__expf(-(dfi[1]/delta)*(dfi[1]/delta));
        float wzi=__expf(-(dfi[2]/delta)*(dfi[2]/delta));
        
        float wxj=__expf(-(dfj[0]/delta)*(dfj[0]/delta));
        float wyj=__expf(-(dfj[1]/delta)*(dfj[1]/delta));
        float wzj=__expf(-(dfj[2]/delta)*(dfj[2]/delta));
        
        float wxk=__expf(-(dfk[0]/delta)*(dfk[0]/delta));
        float wyk=__expf(-(dfk[1]/delta)*(dfk[1]/delta));
        float wzk=__expf(-(dfk[2]/delta)*(dfk[2]/delta));

        
        // this hsould do the trick I think
        
        dftv[idx]=(wx*df[0]+wy*df[1]+wz*df[2])/(sqrt(wx*df[0] *df[0] +wy*df[1] *df[1] +wz*df[2] *df[2])+eps)
        -wzi*dfi[2]/(sqrt(wxi*dfi[0]*dfi[0]+wyi*dfi[1]*dfi[1]+wzi*dfi[2]*dfi[2]) +eps)     // I wish I coudl precompute this, but if I do then Id need to recompute the gradient.
        -wyj*dfj[1]/(sqrt(wxj*dfj[0]*dfj[0]+wyj*dfj[1]*dfj[1]+wzj*dfj[2]*dfj[2]) +eps)
        -wxk*dfk[0]/(sqrt(wxk*dfk[0]*dfk[0]+wyk*dfk[1]*dfk[1]+wzk*dfk[2]*dfk[2]) +eps);
        
    
        return;
        
    }
    
    __device__ void warpReduce(volatile float *sdata, size_t tid) {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    
    __global__ void  reduceNorm2(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        float value=0;
        while (i < n) {
            value=g_idata[i]; //avoid reading twice
            mySum += value*value;
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDART_VERSION >= 9000)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down_sync(0xFFFFFFFF, mySum, offset,32);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    __global__ void  reduceSum(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        // float value=0;
        while (i < n) {
            mySum += g_idata[i];
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDART_VERSION >= 9000)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down_sync(0xFFFFFFFF, mySum, offset,32);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    
    
    
// main function
void aw_pocs_tv(float* img,float* dst,float alpha,const long* image_size, int maxIter,const float delta, const GpuIds& gpuids){
        // Prepare for MultiGPU
        int deviceCount = gpuids.GetLength();
        cudaCheckErrors("Device query fail");
        if (deviceCount == 0) {
            mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPUselect","There are no available device(s) that support CUDA\n");
        }
        //
        // CODE assumes
        // 1.-All available devices are usable by this code
        // 2.-All available devices are equal, they are the same machine (warning thrown)
        // Check the available devices, and if they are the same
        if (!gpuids.AreEqualDevices()) {
            mexWarnMsgIdAndTxt("minimizeDTV:POCS_TV2:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
        }
        int dev;
        
        // We don't know if the devices are being used. lets check that. and only use the amount of memory we need.
        // check free memory
        size_t mem_GPU_global;
        checkFreeMemory(gpuids, &mem_GPU_global);

        
        
        // %5 of free memory should be enough, we have almost no variables in these kernels
        size_t total_pixels              = image_size[0] * image_size[1]  * image_size[2] ;
        size_t mem_slice_image           = sizeof(float)* image_size[0] * image_size[1]  ;
        size_t mem_size_image            = sizeof(float)* total_pixels;
        size_t mem_auxiliary             = sizeof(float)* (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
        
        // Decide how are we handling the distribution of computation
        size_t mem_img_each_GPU;
        
        unsigned int buffer_length=2;
        //Does everything fit in the GPU?
        unsigned int slices_per_split;
        
        // if it is a thin problem (no need to split), just use one GPU
        if (image_size[2]<4){deviceCount=1;}
        
        unsigned int splits=1; // if the number does not fit in an uint, you have more serious trouble than this.
        if(mem_GPU_global> 3*mem_size_image+3*(deviceCount-1)*mem_slice_image*buffer_length+mem_auxiliary) {
            // We only need to split if we have extra GPUs
            slices_per_split=(image_size[2]+deviceCount-1)/deviceCount;
            mem_img_each_GPU=mem_slice_image*((slices_per_split+buffer_length*2));
        }else{
            // As mem_auxiliary is not expected to be a large value (for a 2000^3 image is around 28Mbytes), lets for now assume we need it all
            size_t mem_free=mem_GPU_global-mem_auxiliary;
            
            splits=(unsigned int)(ceil(((float)(3*mem_size_image)/(float)(deviceCount))/mem_free));
            // Now, there is an overhead here, as each splits should have 2 slices more, to account for overlap of images.
            // lets make sure these 2 slices fit, if they do not, add 1 to splits.
            slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits);
            mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            
            // if the new stuff does not fit in the GPU, it means we are in the edge case where adding that extra slice will overflow memory
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                // one more split should do the job, as its an edge case.
                splits++;
                //recompute for later
                slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits); // amount of slices that fit on a GPU. Later we add 2 to these, as we need them for overlap
                mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            }


            // How many EXTRA buffer slices should be able to fit in here??!?!
            // Only do it if there are splits needed. 
            if(splits>1){
                mem_free=mem_GPU_global-(3*mem_img_each_GPU+mem_auxiliary);
                unsigned int extra_buff=(mem_free/mem_slice_image); 
                buffer_length=(extra_buff/2)/3; // we need double whatever this results in, rounded down.
                buffer_length=max(buffer_length,2);// minimum 2
                buffer_length=min(MAX_BUFFER,buffer_length);

                mem_img_each_GPU=mem_slice_image*(slices_per_split+buffer_length*2);
                
            }else{
                buffer_length=2;
            }

            // Assert
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPU","Assertion Failed. Logic behind spliting flawed! Please tell: ander.biguri@gmail.com\n");
            }
        }
        
        
         // Assert
       
        if ((slices_per_split+buffer_length*2)*image_size[0]*image_size[1]* sizeof(float)!= mem_img_each_GPU){
            mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPU","Assertion Failed. Memory needed calculation broken! Please tell: ander.biguri@gmail.com\n");
        }
        
        
        
        
        
        
        float** d_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** d_dimgTV=   (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2aux= (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2=    (float**)malloc(deviceCount*sizeof(float*));
         
        // allocate memory in each GPU
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            
            hipMalloc((void**)&d_image[dev]    , mem_img_each_GPU);
            hipMemset(         d_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&d_dimgTV[dev]   , mem_img_each_GPU);
            hipMemset(         d_dimgTV[dev],0 , mem_img_each_GPU);
            hipMalloc((void**)&d_norm2[dev]    , slices_per_split*mem_slice_image);
            hipMemset(         d_norm2[dev],0  , slices_per_split*mem_slice_image);
            hipMalloc((void**)&d_norm2aux[dev]   , mem_auxiliary);
            hipMemset(         d_norm2aux[dev],0 , mem_auxiliary);
            cudaCheckErrors("Malloc  error");
            
            
        }
       unsigned long long buffer_pixels=buffer_length*image_size[0]*image_size[1];
        float* buffer;
        if(splits>1){
            mexWarnMsgIdAndTxt("minimizeDTV:POCS_TV2:Image_split","Your image can not be fully split between the available GPUs. The computation of minTV will be significantly slowed due to the image size.\nApproximated mathematics turned on for computational speed.");
        }else{
            hipHostMalloc((void**)&buffer,buffer_length*image_size[0]*image_size[1]*sizeof(float));
        }
        
        
        
        // Lets try to make the host memory pinned:
        // We laredy queried the GPU and assuemd they are the same, thus should have the same attributes.
        int isHostRegisterSupported = 0;
#if CUDART_VERSION >= 9020
        hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,gpuids[0]);
#endif
        // splits>2 is completely empirical observation
        if (isHostRegisterSupported & splits>2){
            hipHostRegister(img ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
            hipHostRegister(dst ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
        }
        cudaCheckErrors("Error pinning memory");

        
        
                // Create streams
        int nStream_device=2;
        int nStreams=deviceCount*nStream_device;
        hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));
        
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            for (int i = 0; i < nStream_device; ++i){
                hipStreamCreate(&stream[i+dev*nStream_device]);
            }
        }
        cudaCheckErrors("Stream creation fail");

        
        // For the reduction

        double totalsum_prev;
        double totalsum;
        float sum_curr_spl;
        float * sumnorm2;
        hipHostMalloc((void**)&sumnorm2,deviceCount*sizeof(float));
        
        unsigned int curr_slices;
        unsigned long long curr_pixels;
        size_t linear_idx_start;
        unsigned long long* offset_device=(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* offset_host  =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* bytes_device =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        bool is_first_chunk;
        bool is_last_chunk;
        for(unsigned int i=0;i<maxIter;i+=(buffer_length-1)){
            if(splits>1){
                totalsum_prev=0;
            }
            for(unsigned int sp=0;sp<splits;sp++){
                
                // For each iteration we need to compute all the image. The ordering of these loops
                // need to be like this due to the bounding layers between splits. If more than 1 split is needed
                // for each GPU then there is no other way that taking the entire memory out of GPU and putting it back.
                // If the memory can be shared between GPUs fully without extra splits, then there is an easy way of synchronizing the memory
                
                // Copy image to memory
                for (dev = 0; dev < deviceCount; dev++){
                    curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                    curr_pixels=curr_slices*image_size[0]*image_size[1];
                    linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                    
                    // Check if its the first or last chunck
                    is_last_chunk=!((sp*deviceCount+dev)<deviceCount*splits-1);
                    is_first_chunk=!(sp*deviceCount+dev);
                    
                    // lets compute where we start copyes and how much. This avoids 3 calls to Memcpy
                    offset_device[dev]=buffer_pixels*is_first_chunk;
                    offset_host[dev]=linear_idx_start-buffer_pixels*!is_first_chunk;
                    bytes_device[dev]=curr_pixels+buffer_pixels*!is_first_chunk+buffer_pixels*!is_last_chunk;
                }

                if(i==0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], img+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                    }
                }
                // if we need to split and its not the first iteration, then we need to copy from Host memory the previosu result.
                if (splits>1 & i>0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], dst+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                    }
                }
                cudaCheckErrors("Memcpy failure on multi split");
                
                for(unsigned int ib=0;  (ib<(buffer_length-1)) && ((i+ib)<maxIter);  ib++){
                    
                    // For the gradient
                    dim3 blockGrad(10, 10, 10);
                    dim3 gridGrad((image_size[0]+blockGrad.x-1)/blockGrad.x, (image_size[1]+blockGrad.y-1)/blockGrad.y, (curr_slices+buffer_length*2+blockGrad.z-1)/blockGrad.z);
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // Compute the gradient of the TV norm
                        
                        // I don't understand why I need to store 2 layers to compute correctly with 1 buffer. The bounding checks should
                        // be enough but they are not.
                        gradientTV<<<gridGrad, blockGrad,0,stream[dev*nStream_device]>>>(d_image[dev],d_dimgTV[dev],(long)(curr_slices+buffer_length*2-1), image_size[1],image_size[0],delta);
                        
                    }
                    
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // no need to copy the 2 aux slices here
                        hipStreamSynchronize(stream[dev*nStream_device]);
                        hipMemcpyAsync(d_norm2[dev], d_dimgTV[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice,stream[dev*nStream_device+1]);
                    }
                    
                    
                    // Compute the L2 norm of the gradient. For that, reduction is used.
                    //REDUCE
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
                        
                        hipStreamSynchronize(stream[dev*nStream_device+1]);
                        reduceNorm2 << <dimgridRed, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device]>> >(d_norm2[dev], d_norm2aux[dev], total_pixels);
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;

                        if (dimgridRed > 1) {
                            reduceSum << <1, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device] >> >(d_norm2aux[dev], d_norm2[dev], dimgridRed);
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                        else {
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2aux[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Reduction error");
                    
                    
                    // Accumulate the norm accross devices
                    sum_curr_spl=0;
                    // this is CPU code
                    for (dev = 0; dev < deviceCount; dev++){
                        sum_curr_spl+=sumnorm2[dev];
                    }
                    sum_curr_spl+=0.0000001f; // avoid division by zero
                    
                    // If we have more than one splits, lets use the result from prior calls
                    if(i>0 && splits>1){
                        // this is already stored:
                        //totalsum=totalsum_prev; 
                    }else{
                        totalsum=sum_curr_spl;
                    }
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        //NORMALIZE
                        //in a Tesla, maximum blocks =15 SM * 4 blocks/SM
                        divideArrayScalar  <<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,(float)sqrt(totalsum),total_pixels);
                        //MULTIPLY HYPERPARAMETER
                        multiplyArrayScalar<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,alpha,   total_pixels);
                    }
                     for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Scalar operations error");
                    
                    //SUBSTRACT GRADIENT
                    //////////////////////////////////////////////
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        substractArrays<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_image[dev]+buffer_pixels,d_dimgTV[dev]+buffer_pixels, total_pixels);
                    }
                }

                // Synchronize mathematics, make sure bounding pixels are correct
                 for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                
                if(splits==1){
                    for(dev=0; dev<deviceCount;dev++){
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        if (dev<deviceCount-1){
                            hipSetDevice(gpuids[dev+1]);
                            hipMemcpy(buffer, d_image[dev+1], buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(gpuids[dev]);
                            hipMemcpy(d_image[dev]+total_pixels+buffer_pixels,buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice); 
                        }
                        hipDeviceSynchronize();
                        if (dev>0){
                            hipSetDevice(gpuids[dev-1]);
                            hipMemcpyAsync(buffer, d_image[dev-1]+total_pixels+buffer_pixels, buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(gpuids[dev]);
                            hipMemcpyAsync(d_image[dev],buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                    }
                }else{
                    
                    // We need to take it out :(
                    for(dev=0; dev<deviceCount;dev++){
                        hipSetDevice(gpuids[dev]);
                        
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        hipMemcpyAsync(&dst[linear_idx_start], d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                    }
                }
                
                for (dev = 0; dev < deviceCount; dev++){
                    hipSetDevice(gpuids[dev]);
                    hipDeviceSynchronize();
                }
                cudaCheckErrors("Memory gather error");

                totalsum_prev+=sum_curr_spl;
            }
            totalsum=totalsum_prev;
        }
        // If there has not been splits, we still have data in memory
        if(splits==1){
            for(dev=0; dev<deviceCount;dev++){
                hipSetDevice(gpuids[dev]);
                
                curr_slices=((dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*dev;
                total_pixels=curr_slices*image_size[0]*image_size[1];
                hipMemcpy(dst+slices_per_split*image_size[0]*image_size[1]*dev, d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost);
            }
        }
        cudaCheckErrors("Copy result back");
        
        for(dev=0; dev<deviceCount;dev++){
            hipSetDevice(gpuids[dev]);
            hipFree(d_image[dev]);
            hipFree(d_norm2aux[dev]);
            hipFree(d_dimgTV[dev]);
            hipFree(d_norm2[dev]);
        }
        if (splits==1){
            hipHostFree(buffer);
        }
        
        if (isHostRegisterSupported& splits>2){
            hipHostUnregister(img);
            hipHostUnregister(dst);
        }
        for (int i = 0; i < nStreams; ++i)
           hipStreamDestroy(stream[i]) ;
        cudaCheckErrors("Memory free");
//         hipDeviceReset();
    }

void dtv_pocs(float* img,float* dst,float alpha,const long* image_size, int maxIter, int APGMiter, float mu, const float delta, const GpuIds& gpuids){
        // Prepare for MultiGPU
        int deviceCount = gpuids.GetLength();
        cudaCheckErrors("Device query fail");
        if (deviceCount == 0) {
            mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPUselect","There are no available device(s) that support CUDA\n");
        }
        //
        // CODE assumes
        // 1.-All available devices are usable by this code
        // 2.-All available devices are equal, they are the same machine (warning thrown)
        // Check the available devices, and if they are the same
        if (!gpuids.AreEqualDevices()) {
            mexWarnMsgIdAndTxt("minimizeDTV:POCS_TV2:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
        }
        int dev;
        
        // We don't know if the devices are being used. lets check that. and only use the amount of memory we need.
        // check free memory
        size_t mem_GPU_global;
        checkFreeMemory(gpuids, &mem_GPU_global);

        
        
        // %5 of free memory should be enough, we have almost no variables in these kernels
        size_t total_pixels              = image_size[0] * image_size[1]  * image_size[2] ;
        size_t mem_slice_image           = sizeof(float)* image_size[0] * image_size[1]  ;
        size_t mem_size_image            = sizeof(float)* total_pixels;
        size_t mem_auxiliary             = sizeof(float)* (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
        
        // Decide how are we handling the distribution of computation
        size_t mem_img_each_GPU;
        
        unsigned int buffer_length=2;
        //Does everything fit in the GPU?
        unsigned int slices_per_split;
        
        // if it is a thin problem (no need to split), just use one GPU
        if (image_size[2]<4){deviceCount=1;}
        
        unsigned int splits=1; // if the number does not fit in an uint, you have more serious trouble than this.
        if(mem_GPU_global> 3*mem_size_image+3*(deviceCount-1)*mem_slice_image*buffer_length+mem_auxiliary) {
            // We only need to split if we have extra GPUs
            slices_per_split=(image_size[2]+deviceCount-1)/deviceCount;
            mem_img_each_GPU=mem_slice_image*((slices_per_split+buffer_length*2));
        }else{
            // As mem_auxiliary is not expected to be a large value (for a 2000^3 image is around 28Mbytes), lets for now assume we need it all
            size_t mem_free=mem_GPU_global-mem_auxiliary;
            
            splits=(unsigned int)(ceil(((float)(3*mem_size_image)/(float)(deviceCount))/mem_free));
            // Now, there is an overhead here, as each splits should have 2 slices more, to account for overlap of images.
            // lets make sure these 2 slices fit, if they do not, add 1 to splits.
            slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits);
            mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            
            // if the new stuff does not fit in the GPU, it means we are in the edge case where adding that extra slice will overflow memory
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                // one more split should do the job, as its an edge case.
                splits++;
                //recompute for later
                slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits); // amount of slices that fit on a GPU. Later we add 2 to these, as we need them for overlap
                mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            }


            // How many EXTRA buffer slices should be able to fit in here??!?!
            // Only do it if there are splits needed. 
            if(splits>1){
                mem_free=mem_GPU_global-(3*mem_img_each_GPU+mem_auxiliary);
                unsigned int extra_buff=(mem_free/mem_slice_image); 
                buffer_length=(extra_buff/2)/3; // we need double whatever this results in, rounded down.
                buffer_length=max(buffer_length,2);// minimum 2
                buffer_length=min(MAX_BUFFER,buffer_length);

                mem_img_each_GPU=mem_slice_image*(slices_per_split+buffer_length*2);
                
            }else{
                buffer_length=2;
            }

            // Assert
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPU","Assertion Failed. Logic behind spliting flawed! Please tell: ander.biguri@gmail.com\n");
            }
        }
        
        
         // Assert
       
        if ((slices_per_split+buffer_length*2)*image_size[0]*image_size[1]* sizeof(float)!= mem_img_each_GPU){
            mexErrMsgIdAndTxt("minimizeDTV:POCS_TV2:GPU","Assertion Failed. Memory needed calculation broken! Please tell: ander.biguri@gmail.com\n");
        }
        

        float** d_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** p1_=    (float**)malloc(deviceCount*sizeof(float*)); 
        float** p2_=    (float**)malloc(deviceCount*sizeof(float*)); 
        float** p1_image=    (float**)malloc(deviceCount*sizeof(float*)); 
        float** p2_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v11_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v21_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v31_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v12_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v22_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** v32_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** d_dimgTV=   (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2aux= (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2=    (float**)malloc(deviceCount*sizeof(float*));
         
        // allocate memory in each GPU
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            
            hipMalloc((void**)&d_image[dev]    , mem_img_each_GPU);
            hipMemset(         d_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&p1_[dev]    , mem_img_each_GPU);
            hipMemset(         p1_[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&p2_[dev]    , mem_img_each_GPU);
            hipMemset(         p2_[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&p1_image[dev]    , mem_img_each_GPU);
            hipMemset(         p1_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&p2_image[dev]    , mem_img_each_GPU);
            hipMemset(         p2_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v11_image[dev]    , mem_img_each_GPU);
            hipMemset(         v11_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v21_image[dev]    , mem_img_each_GPU);
            hipMemset(         v21_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v31_image[dev]    , mem_img_each_GPU);
            hipMemset(         v31_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v12_image[dev]    , mem_img_each_GPU);
            hipMemset(         v12_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v22_image[dev]    , mem_img_each_GPU);
            hipMemset(         v22_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&v32_image[dev]    , mem_img_each_GPU);
            hipMemset(         v32_image[dev],0  , mem_img_each_GPU);
            hipMalloc((void**)&d_dimgTV[dev]   , mem_img_each_GPU);
            hipMemset(         d_dimgTV[dev],0 , mem_img_each_GPU);
            hipMalloc((void**)&d_norm2[dev]    , slices_per_split*mem_slice_image);
            hipMemset(         d_norm2[dev],0  , slices_per_split*mem_slice_image);
            hipMalloc((void**)&d_norm2aux[dev]   , mem_auxiliary);
            hipMemset(         d_norm2aux[dev],0 , mem_auxiliary);
            cudaCheckErrors("Malloc  error");
            
            
        }
       unsigned long long buffer_pixels=buffer_length*image_size[0]*image_size[1];
        float* buffer;
        if(splits>1){
            mexWarnMsgIdAndTxt("minimizeDTV:POCS_TV2:Image_split","Your image can not be fully split between the available GPUs. The computation of minTV will be significantly slowed due to the image size.\nApproximated mathematics turned on for computational speed.");
        }else{
            hipHostMalloc((void**)&buffer,buffer_length*image_size[0]*image_size[1]*sizeof(float));
        }
        
        
        
        // Lets try to make the host memory pinned:
        // We laredy queried the GPU and assuemd they are the same, thus should have the same attributes.
        int isHostRegisterSupported = 0;
#if CUDART_VERSION >= 9020
        hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,gpuids[0]);
#endif
        // splits>2 is completely empirical observation
        if (isHostRegisterSupported & splits>2){
            hipHostRegister(img ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
            hipHostRegister(dst ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
        }
        cudaCheckErrors("Error pinning memory");

        
        
                // Create streams
        int nStream_device=2;
        int nStreams=deviceCount*nStream_device;
        hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));
        
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            for (int i = 0; i < nStream_device; ++i){
                hipStreamCreate(&stream[i+dev*nStream_device]);
            }
        }
        cudaCheckErrors("Stream creation fail");

        
        // For the reduction

        double totalsum_prev;
        double totalsum;
        float sum_curr_spl;
        float * sumnorm2;
        hipHostMalloc((void**)&sumnorm2,deviceCount*sizeof(float));
        
        unsigned int curr_slices;
        unsigned long long curr_pixels;
        size_t linear_idx_start;
        unsigned long long* offset_device=(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* offset_host  =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* bytes_device =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        bool is_first_chunk;
        bool is_last_chunk;
        for(unsigned int i=0;i<maxIter;i+=(buffer_length-1)){
            if(splits>1){
                totalsum_prev=0;
            }
            for(unsigned int sp=0;sp<splits;sp++){
                
                // For each iteration we need to compute all the image. The ordering of these loops
                // need to be like this due to the bounding layers between splits. If more than 1 split is needed
                // for each GPU then there is no other way that taking the entire memory out of GPU and putting it back.
                // If the memory can be shared between GPUs fully without extra splits, then there is an easy way of synchronizing the memory
                
                // Copy image to memory
                for (dev = 0; dev < deviceCount; dev++){
                    curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                    curr_pixels=curr_slices*image_size[0]*image_size[1];
                    linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                    
                    // Check if its the first or last chunck
                    is_last_chunk=!((sp*deviceCount+dev)<deviceCount*splits-1);
                    is_first_chunk=!(sp*deviceCount+dev);
                    
                    // lets compute where we start copyes and how much. This avoids 3 calls to Memcpy
                    offset_device[dev]=buffer_pixels*is_first_chunk;
                    offset_host[dev]=linear_idx_start-buffer_pixels*!is_first_chunk;
                    bytes_device[dev]=curr_pixels+buffer_pixels*!is_first_chunk+buffer_pixels*!is_last_chunk;
                }

                if(i==0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], img+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                    }
                }
                // if we need to split and its not the first iteration, then we need to copy from Host memory the previosu result.
                if (splits>1 & i>0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], dst+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                    }
                }
                cudaCheckErrors("Memcpy failure on multi split");
                
                for(unsigned int ib=0;  (ib<(buffer_length-1)) && ((i+ib)<maxIter);  ib++){
                    
                    dim3 blockGrad(10, 10, 10);
                    dim3 gridGrad((image_size[0]+blockGrad.x-1)/blockGrad.x, (image_size[1]+blockGrad.y-1)/blockGrad.y, (curr_slices+buffer_length*2+blockGrad.z-1)/blockGrad.z);
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        
                        //COMPUTE p1_image and p2_image using ADMM then use them to compute the dTV gradient
                        APGM<<<gridGrad, blockGrad,0,stream[dev*nStream_device]>>>(d_image[dev],p1_image[dev],p2_image[dev],v11_image[dev],v21_image[dev],v31_image[dev],v12_image[dev],v22_image[dev],v32_image[dev],(long)(curr_slices+buffer_length*2-1), image_size[1],image_size[0],delta,mu,APGMiter);

                    }

                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // no need to copy the 2 aux slices here
                        hipStreamSynchronize(stream[dev*nStream_device]);
                        hipMemcpyAsync(p1_[dev], p1_image[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice,stream[dev*nStream_device+1]);
                        hipMemcpyAsync(p2_[dev], p2_image[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice,stream[dev*nStream_device+1]);
                    }
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        
                        // Compute the gradient of the TV norm
                        // I don't understand why I need to store 2 layers to compute correctly with 1 buffer. The bounding checks should
                        // be enough but they are not.
                        gradientDTV<<<gridGrad, blockGrad,0,stream[dev*nStream_device]>>>(p1_[dev],p2_[dev],d_dimgTV[dev],(long)(curr_slices+buffer_length*2-1), image_size[1],image_size[0],delta);
                        
                    }
                    
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // no need to copy the 2 aux slices here
                        hipStreamSynchronize(stream[dev*nStream_device]);
                        hipMemcpyAsync(d_norm2[dev], d_dimgTV[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice,stream[dev*nStream_device+1]);
                    }
                    
                    
                    // Compute the L2 norm of the gradient. For that, reduction is used.
                    //REDUCE
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
                        
                        hipStreamSynchronize(stream[dev*nStream_device+1]);
                        reduceNorm2 << <dimgridRed, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device]>> >(d_norm2[dev], d_norm2aux[dev], total_pixels);
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;

                        if (dimgridRed > 1) {
                            reduceSum << <1, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device] >> >(d_norm2aux[dev], d_norm2[dev], dimgridRed);
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                        else {
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2aux[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Reduction error");
                    
                    
                    // Accumulate the norm accross devices
                    sum_curr_spl=0;
                    // this is CPU code
                    for (dev = 0; dev < deviceCount; dev++){
                        sum_curr_spl+=sumnorm2[dev];
                    }
                    sum_curr_spl+=0.0000001f; // avoid division by zero
                    
                    // If we have more than one splits, lets use the result from prior calls
                    if(i>0 && splits>1){
                        // this is already stored:
                        //totalsum=totalsum_prev; 
                    }else{
                        totalsum=sum_curr_spl;
                    }
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        //NORMALIZE
                        //in a Tesla, maximum blocks =15 SM * 4 blocks/SM
                        divideArrayScalar  <<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,(float)sqrt(totalsum),total_pixels);
                        //MULTIPLY HYPERPARAMETER
                        multiplyArrayScalar<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,alpha,   total_pixels);
                    }
                     for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Scalar operations error");
                    
                    //SUBSTRACT GRADIENT
                    //////////////////////////////////////////////
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        substractArrays<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_image[dev]+buffer_pixels,d_dimgTV[dev]+buffer_pixels, total_pixels);
                    }
                }

                // Synchronize mathematics, make sure bounding pixels are correct
                 for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(gpuids[dev]);
                        hipDeviceSynchronize();
                     }
                
                if(splits==1){
                    for(dev=0; dev<deviceCount;dev++){
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        if (dev<deviceCount-1){
                            hipSetDevice(gpuids[dev+1]);
                            hipMemcpy(buffer, d_image[dev+1], buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(gpuids[dev]);
                            hipMemcpy(d_image[dev]+total_pixels+buffer_pixels,buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice); 
                        }
                        hipDeviceSynchronize();
                        if (dev>0){
                            hipSetDevice(gpuids[dev-1]);
                            hipMemcpyAsync(buffer, d_image[dev-1]+total_pixels+buffer_pixels, buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(gpuids[dev]);
                            hipMemcpyAsync(d_image[dev],buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                    }
                }else{
                    
                    // We need to take it out :(
                    for(dev=0; dev<deviceCount;dev++){
                        hipSetDevice(gpuids[dev]);
                        
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        hipMemcpyAsync(&dst[linear_idx_start], d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                    }
                }
                
                for (dev = 0; dev < deviceCount; dev++){
                    hipSetDevice(gpuids[dev]);
                    hipDeviceSynchronize();
                }
                cudaCheckErrors("Memory gather error");

                totalsum_prev+=sum_curr_spl;
            }
            totalsum=totalsum_prev;
        }
        // If there has not been splits, we still have data in memory
        if(splits==1){
            for(dev=0; dev<deviceCount;dev++){
                hipSetDevice(gpuids[dev]);
                
                curr_slices=((dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*dev;
                total_pixels=curr_slices*image_size[0]*image_size[1];
                hipMemcpy(dst+slices_per_split*image_size[0]*image_size[1]*dev, d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost);
            }
        }
        cudaCheckErrors("Copy result back");
        
        for(dev=0; dev<deviceCount;dev++){
            hipSetDevice(gpuids[dev]);
            hipFree(d_image[dev]);
            hipFree(d_norm2aux[dev]);
            hipFree(d_dimgTV[dev]);
            hipFree(d_norm2[dev]);
        }
        if (splits==1){
            hipHostFree(buffer);
        }
        
        if (isHostRegisterSupported& splits>2){
            hipHostUnregister(img);
            hipHostUnregister(dst);
        }
        for (int i = 0; i < nStreams; ++i)
           hipStreamDestroy(stream[i]) ;
        cudaCheckErrors("Memory free");
//         hipDeviceReset();
    }
        
void checkFreeMemory(const GpuIds& gpuids, size_t *mem_GPU_global){
        size_t memfree;
        size_t memtotal;
        const int deviceCount = gpuids.GetLength();
        for (int dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            hipMemGetInfo(&memfree,&memtotal);
            if(dev==0) *mem_GPU_global=memfree;
            if(memfree<memtotal/2){
                mexErrMsgIdAndTxt("tvDenoise:tvdenoising:GPU","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
            }
            cudaCheckErrors("Check mem error");
            
            *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
        }
        *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
        
        //*mem_GPU_global= insert your known number here, in bytes.
}
